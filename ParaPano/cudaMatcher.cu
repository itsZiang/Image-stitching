#include "hip/hip_runtime.h"
#include <string>
#include <math.h>
#include <stdio.h>
#include <vector>
 #include ""

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaMatcher.h"

#define NUM_THREADS_PER_BLOCK 512

using namespace std;

CudaMatcher::CudaMatcher() {
    cudaDesc1 = NULL;
    cudaDesc2 = NULL;
    cuda_ratios = NULL;
    cuda_match_indices = NULL;
}

CudaMatcher::~CudaMatcher() {
    // if (cudaDesc1) {
    //     // free image data on CUDA
    //     hipFree(cudaDesc1);
    //     hipFree(cudaDesc2);
    //     hipFree(cuda_ratios);
    //     hipFree(cuda_match_indices);
    // }
}

Descriptor* copyDescriptorToDevice(vector<Descriptor>& desc) {

    int num_desc = desc.size();

    // convert vector to array to be used on Cuda Device
    // Descriptor* tempDesc = (Descriptor*) malloc(sizeof(Descriptor) * num_desc);
    Descriptor* tempDesc = new Descriptor[num_desc];
    for (int i = 0; i < num_desc; i++) {
        tempDesc[i] = desc[i];
    }

    Descriptor* cudaDesc;
    hipMalloc((void**)&cudaDesc, sizeof(Descriptor) * num_desc);
    hipMemcpy(cudaDesc, tempDesc, sizeof(Descriptor) * num_desc, hipMemcpyHostToDevice);

    return cudaDesc;
}

void
CudaMatcher::setup(vector<Descriptor> desc1, vector<Descriptor> desc2) {
    // printCudaInfo();
    num_desc1 = desc1.size();
    num_desc2 = desc2.size();
    cudaDesc1 = copyDescriptorToDevice(desc1);
    cudaDesc2 = copyDescriptorToDevice(desc2);
}


__device__ __inline__ int countOneBits(uint64_t i) {
    i = i - ((i >> 1) & 0x5555555555555555);
    i = (i & 0x3333333333333333) + ((i >> 2) & 0x3333333333333333);
    return (((i + (i >> 4)) & 0xF0F0F0F0F0F0F0F) * 0x101010101010101) >> 56;
    // int count = 0;
    // while(i) {
    //   i &= (i - 1);
    //   count++;
    // }
    // return count;
}


__device__ int hammingDistance(Descriptor d1, Descriptor d2) {
    int dist = 0;
    // Performs distance calculation with 4 uint64_t descriptor
    dist += countOneBits(d1.num0 ^ d2.num0);
    dist += countOneBits(d1.num1 ^ d2.num1);
    dist += countOneBits(d1.num2 ^ d2.num2);
    dist += countOneBits(d1.num3 ^ d2.num3);

    return dist;
}

__global__ void match(Descriptor* cudaDesc1, Descriptor* cudaDesc2,
    int num_desc1, int num_desc2, float* ratios, int* match_indices) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Invalid index
    if (idx >= num_desc1) {
        return;
    }
    Descriptor d1 = cudaDesc1[idx];

    int min = INT_MAX;
    int second_min = INT_MAX;
    int min_idx = -1;
    for (int i = 0; i < num_desc2; i++) {
        int dist = hammingDistance(d1, cudaDesc2[i]);
        if (dist < min) {
            second_min = min;
            min = dist;
            min_idx = i;
        } else if (dist < second_min) {
            second_min = dist;
        }
    }

    float r = second_min == 0 ? 1 : float(min) / second_min;
    
    match_indices[idx] = min_idx;
    ratios[idx] = r;
}

void
CudaMatcher::getMatchResult(float* ratios, int* match_indices) {

    hipMemcpy(ratios, cuda_ratios, sizeof(float) * num_desc1,
        hipMemcpyDeviceToHost);

    hipMemcpy(match_indices, cuda_match_indices, sizeof(int) * num_desc1,
        hipMemcpyDeviceToHost);
}



MatchResult
CudaMatcher::findMatch() {
    float ratio_threshold = 0.8;
    float* ratios = (float*) malloc(sizeof(float) * num_desc1);
    int* match_indices = (int*) malloc(sizeof(int) * num_desc1);

    hipError_t err = hipMalloc(&cuda_ratios, sizeof(float) * num_desc1);
    err = hipMalloc(&cuda_match_indices, sizeof(int) * num_desc1);
    if (err) throw err;

    // Spawn CUDA threads
    int num_blocks = (num_desc1 + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;

    match<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(cudaDesc1, cudaDesc2, 
        num_desc1, num_desc2, cuda_ratios, cuda_match_indices);
    hipDeviceSynchronize();

    getMatchResult(ratios, match_indices);

    MatchResult match_result;
    for (int i = 0; i < num_desc1; i++) {
        if (ratios[i] < ratio_threshold) {
            match_result.indices1.push_back(i);
            match_result.indices2.push_back(match_indices[i]);
        }
    }

    free(ratios);
    free(match_indices);
    return match_result;
}
