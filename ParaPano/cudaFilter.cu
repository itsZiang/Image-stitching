#include "hip/hip_runtime.h"
#include <string>
#include <math.h>
#include <stdio.h>
#include <vector>
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaFilter.h"

#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 16

using namespace std;

CudaFilterer::CudaFilterer() {
    gaussian_pyramid = NULL; // result on CPU
    cudaImageData = NULL;
    cudaGaussianPyramid = NULL;
    imageWidth = 0;
    imageHeight = 0;
    numLevels = 0;
}

CudaFilterer::~CudaFilterer() {
    if (cudaImageData) {
        // free image data on CUDA
        hipFree(cudaImageData);
    }
}

void
printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

void
CudaFilterer::allocHostGaussianPyramid(int width, int height, int num_levels) {
    gaussian_pyramid = new float*[num_levels];
    for (int i = 0; i < num_levels; i++) {
        gaussian_pyramid[i] = new float[width * height];
    }
}

void
CudaFilterer::allocDeviceGaussianPyramid(int width, int height) {
    hipMalloc(&cudaGaussianPyramid, sizeof(float) * width * height);
}

void
CudaFilterer::getGaussianPyramid(int i) {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    hipMemcpy(gaussian_pyramid[i],
        cudaGaussianPyramid,
        sizeof(float) * imageWidth * imageHeight,
        hipMemcpyDeviceToHost);
}

void
CudaFilterer::setup(float* img, int h, int w) {

    // printCudaInfo();

    // set parameters
    imageHeight = h;
    imageWidth = w;

    // copy image data from host to device
    hipMalloc(&cudaImageData, sizeof(float) * w * h);
    hipMemcpy(cudaImageData, img, sizeof(float) * w * h, hipMemcpyHostToDevice);
}


// create a normalized gaussian filter of height h and width w
float*
createHostGaussianFilter(const int fh, const int fw, float sigma) {
    float* gaussianFilter = new float[fh * fw];
    float sum = 0.0;
    int centerX = fw/2;
    int centerY = fh/2;
    for (int i = 0; i < fh; i++) {
        for (int j = 0; j < fw; j++) {
            int x = j - centerX;
            int y = i - centerY;
            float e = -(x*x + y*y) / (2 * sigma * sigma);
            gaussianFilter[i * fw + j] = exp(e) / (2 * M_PI * sigma * sigma);
            sum += gaussianFilter[i * fw + j];
        }
    }

    // normalize
    for (int i = 0; i < fh; i++) {
        for (int j = 0; j < fw; j++) {
            gaussianFilter[i * fw + j] /= sum;
        }
    }
    return gaussianFilter;
}

__device__ __inline__ bool
inBound(int r, int c, int h, int w) {
    return r >= 0 && r < h && c >= 0 && c < w;
}

/*
 * kernel function
 */
__global__ void
applyGaussianFilter(const float* img_ptr, int h, int w, 
                    float* cudaFilter, int fsize, float* cudaGaussianPyramid) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float weightedSum = 0.0;
    int fhHalf = fsize / 2;
    int fwHalf = fsize / 2;

    for (int ii = -fhHalf; ii < fsize - fhHalf; ii++) {
        for (int jj = -fwHalf; jj < fsize - fwHalf; jj++) {
            int r = row + ii;
            int c = col + jj;
            float imVal = inBound(r, c, h, w) ? img_ptr[r * w + c] : 0;
            weightedSum += imVal * cudaFilter[(ii+fhHalf)*fsize + (jj+fwHalf)];
        }
    }
    cudaGaussianPyramid[row * w + col] = weightedSum;
}
 
float**
CudaFilterer::createGaussianPyramid(float sigma0, float k, const int* levels,
                                    int num_levels) {

    numLevels = num_levels;

    // allocate host memory
    allocHostGaussianPyramid(imageWidth, imageHeight, num_levels);

    allocDeviceGaussianPyramid(imageWidth, imageHeight);

    for (int i = 0; i < num_levels; i++) {
        float sigma = sigma0 * pow(k, levels[i]);
        int fsize = floor(3 * sigma * 2) + 1;
        float* filter = createHostGaussianFilter(fsize, fsize, sigma);

        // copy filter to CUDA memory
        float* cudaFilter;
        hipMalloc(&cudaFilter, sizeof(float) * fsize * fsize);
        hipMemcpy(cudaFilter, filter, sizeof(float) * fsize * fsize,
                   hipMemcpyHostToDevice);

        // Spawn CUDA threads
        dim3 gridDim(imageWidth / BLOCK_WIDTH, imageHeight / BLOCK_HEIGHT);
        dim3 blockDim(BLOCK_WIDTH, BLOCK_HEIGHT);

        applyGaussianFilter<<< gridDim, blockDim>>>(cudaImageData, imageHeight,
                        imageWidth, cudaFilter, fsize, cudaGaussianPyramid);

        // Transfer the ith pyramid from device to host's gaussian_pyramid
        getGaussianPyramid(i);

        // clean up memory
        delete[] filter;
        hipFree(cudaFilter);
    }
    hipFree(cudaGaussianPyramid);

    return gaussian_pyramid;
}
